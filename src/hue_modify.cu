#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <png.h>

// Funcao que aplica a matriz de transformacao A
// ao pixel px = (r, g, b)
// (new_r, new_g, new_b)' = A * (r, g, b)'
__host__ __device__ void modify_pixel(png_bytep px, double *A) {
    double r = px[0] / 255.0;
    double g = px[1] / 255.0;
    double b = px[2] / 255.0;

    double new_r = A[0] * r + A[1] * g + A[2] * b;
    double new_g = A[3] * r + A[4] * g + A[5] * b;
    double new_b = A[6] * r + A[7] * g + A[8] * b;

    new_r = fmin(fmax(new_r, 0.0), 1.0);
    new_g = fmin(fmax(new_g, 0.0), 1.0);
    new_b = fmin(fmax(new_b, 0.0), 1.0);

    px[0] = (png_byte) round(new_r * 255.0);
    px[1] = (png_byte) round(new_g * 255.0);
    px[2] = (png_byte) round(new_b * 255.0);
}

// Altera a matiz (hue) de uma imagem sequencialmente
void modify_hue_seq(png_bytep image, int width, int height, double hue_diff) {
    double c = cos(2 * M_PI * hue_diff);
    double s = sin(2 * M_PI * hue_diff);
    double one_third = 1.0 / 3.0;
    double sqrt_third = sqrt(one_third);

    // Matriz A compoe as operacoes de
    // conversao de RGB para HSV, mudanca de hue,
    // e conversao de HSV de volta para RGB
    // (new_r, new_g, new_b)' = A * (r, g, b)'
    // https://stackoverflow.com/questions/8507885/shift-hue-of-an-rgb-color

    double a11 = c + one_third * (1.0 - c);
    double a12 = one_third * (1.0 - c) - sqrt_third * s;
    double a13 = one_third * (1.0 - c) + sqrt_third * s;
    double a21 = a13; double a22 = a11; double a23 = a12;
    double a31 = a12; double a32 = a13; double a33 = a11;

    double A[9] = {a11, a12, a13, a21, a22, a23, a31, a32, a33};

    for (int i = 0; i < height; i++) {
        png_bytep row = &(image[i * width * 3]);
        for (int j = 0; j < width; j++) {
            png_bytep px = &(row[j * 3]);
            modify_pixel(px, A);
        }
    }
}

// Funcao auxiliar para identificar erros CUDA
void checkErrors(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s [Erro CUDA: %s]\n",
                msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Kernel CUDA para alteracao do hue
// Voce deve modificar essa funcao no EP3
__global__ void modify_hue_kernel(png_bytep d_image, int width, int height, double *A) {
    // SEU CODIGO DO EP3 AQUI

    //calcula coordenadas do pixel para a thread CUDA
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    //caso a thread esteja nos limites válidos da imagem
    if(idx < width && idy < height){
        //indice do pixel no vetor 'd_image', 3 canais RBG
        int pixel_idx = (idy * width + idx) * 3;
        //valores são lidos e normalizados no  intervalo [0, 1], / 255
        double r = d_image[pixel_idx] / 255.0;
        double g = d_image[pixel_idx + 1] / 255.0;
        double b = d_image[pixel_idx + 2] / 255.0;

        //novos valores de RGB

        double novo_r = r * d_A[0] + g * d_A[1] + b * d_A[2];
        double novo_g = r * d_A[3] + g * d_A[4] + b * d_A[5];
        double novo_b = r * d_A[6] + g * d_A[7] + b * d_A[8];

        //min e max garantem que os novos valores estejam dentro do intervalo [0, 1]
        novo_r = fmin(fmax(novo_r, 0.0), 1.0);
        novo_g = fmin(fmax(novo_g, 0.0), 1.0);
        novo_b = fmin(fmax(novo_b, 0.0), 1.0);

        //após garantir que os valores estão no intervalo, os valores são normalizados de volta para o intervalo [0, 255] e arredondando para o valor inteiro mais proximo e armazenados na imagem de saída (d_image)
        d_image[pixel_idx] = (png_byte)round(novo_r * 255.0);
        d_image[pixel_idx + 1] = (png_byte)round(novo_g * 255.0);
        d_image[pixel_idx + 2] = (png_byte)round(novo_b * 255.0);
    }
}



// Altera a matiz (hue) de uma imagem em paralelo
// Voce deve modificar essa funcao no EP3
void modify_hue(png_bytep h_image,
                int width,
                int height,
                size_t image_size,
                double hue_diff) {
    // SEU CODIGO DO EP3 AQUI

    // Voce deve completar os ... com os argumentos corretos e
    // indicar dimensoes apropriadas para o grid e os blocos
    // (blocos por grid e threads por bloco)

    // As mensagens nas chamadas de checkErrors, usadas pra debug,
    // sao uma "dica" do que deve ser feito em cada chamada a funcoes CUDA

    // hipMalloc(...);
    // checkErrors(hipGetLastError(), "Alocacao da matriz A no device");

    // hipMemcpy(...);
    // checkErrors(hipGetLastError(), "Copia da matriz A para o device");

    // hipMalloc(...);
    // checkErrors(hipGetLastError(), "Alocacao da imagem no device");

    // hipMemcpy(...);
    // checkErrors(hipGetLastError(), "Copia da imagem para o device");

    // // Determinar as dimensoes adequadas aqui
    // dim3 dim_block(1, 1);
    // dim3 dim_grid(1, 1);

    // modify_hue_kernel<<<dim_grid, dim_block>>>
    //     (...);
    // checkErrors(hipGetLastError(), "Lançamento do kernel");

    // hipMemcpy(...);
    // checkErrors(hipGetLastError(), "Copia da imagem para o host");

    // hipFree(...);
    // hipFree(...);
}

// Le imagem png de um arquivo de entrada para a memoria
void read_png_image(const char *filename,
                    png_bytep *image,
                    int *width,
                    int *height,
                    size_t *image_size) {
    FILE *fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Erro ao ler o arquivo de entrada %s\n", filename);
        exit(EXIT_FAILURE);
    }

    png_structp png = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png) {
        fprintf(stderr, "Erro ao criar PNG read struct \n");
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_infop info = png_create_info_struct(png);
    if (!info) {
        fprintf(stderr, "Erro ao criar PNG info struct \n");
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    // Em caso de erro nas funcoes da libpng,
    // programa "pula" para este ponto de execucao
    if (setjmp(png_jmpbuf(png))) {
        fprintf(stderr, "Erro ao ler imagem PNG \n");
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_init_io(png, fp);
    png_read_info(png, info);

    *width = png_get_image_width(png, info);
    *height = png_get_image_height(png, info);
    png_byte color_type = png_get_color_type(png, info);
    png_byte bit_depth = png_get_bit_depth(png, info);

    // Verifica se imagem png possui o formato apropriado
    if ((color_type != PNG_COLOR_TYPE_RGB && color_type != PNG_COLOR_TYPE_GRAY)
        || bit_depth != 8) {
        printf("Formato PNG nao suportado, deve ser 8-bit RGB ou grayscale\n");
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_read_update_info(png, info);

    // Alocacao de memoria para imagem e ponteiros para as linhas
    *image_size = png_get_rowbytes(png, info) * (*height);
    *image = (png_bytep) malloc(*image_size);

    png_bytep *row_pointers = (png_bytep *) malloc(sizeof(png_bytep) * (*height));
    for (int i = 0; i < *height; i++) {
        row_pointers[i] = *image + i * png_get_rowbytes(png, info);
    }

    // Leitura da imagem para a memoria
    png_read_image(png, row_pointers);

    // Finalizacao da leitura
    png_destroy_read_struct(&png, &info, NULL);
    fclose(fp);
    free(row_pointers);
}

// Escreve imagem png da memoria para um arquivo de saida
void write_png_image(const char *filename,
                     png_bytep image,
                     int width,
                     int height) {
    FILE *fp = fopen(filename, "wb");
    if (!fp) {
        fprintf(stderr, "Erro ao criar o arquivo de saida %s\n", filename);
        exit(EXIT_FAILURE);
    }

    png_structp png = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png) {
        fprintf(stderr, "Erro ao criar PNG write struct \n");
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_infop info = png_create_info_struct(png);
    if (!info) {
        fprintf(stderr, "Erro ao criar PNG info struct.\n");
        png_destroy_write_struct(&png, &info);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    // Em caso de erro nas funcoes da libpng,
    // programa "pula" para este ponto de execucao
    if (setjmp(png_jmpbuf(png))) {
        printf("Erro ao escrever imagem PNG \n");
        png_destroy_write_struct(&png, &info);
        fclose(fp);
        return;
    }

    png_init_io(png, fp);

    // Configura o formato da imagem a ser criada
    png_set_IHDR(
        png, info, width, height, 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
        PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT
    );

    png_write_info(png, info);

    // Criacao de ponteiros para as linhas
    png_bytep row_pointers[height];
    for (int i = 0; i < height; i++) {
        row_pointers[i] = &(image[i * width * 3]);
    }

    // Escrita da imagem a partir da memoria
    png_write_image(png, row_pointers);
    png_write_end(png, NULL);

    // Finalizacao da escrita
    png_destroy_write_struct(&png, &info);
    fclose(fp);
}

int main(int argc, char *argv[]) {
    png_bytep image;
    int width, height;
    size_t image_size;

    // Leitura e validacao dos parametros de entrada
    if (argc != 4) {
        printf("Uso: ./hue_modify <input_file> <output_file> <hue_diff>\n");
        printf("0.0 <= hue_diff <= 1.0\n");
        exit(EXIT_FAILURE);
    }

    double hue_diff;
    int ret = sscanf(argv[3], "%lf", &hue_diff);
    if (ret == 0 || ret == EOF) {
        fprintf(stderr, "Erro ao ler hue_diff\n");
        exit(EXIT_FAILURE);
    }

    if (hue_diff < 0.0 || hue_diff > 1.0) {
        fprintf(stderr, "hue_diff deve ser entre 0.0 e 1.0\n");
        exit(EXIT_FAILURE);
    }

    // Leitura da imagem para memoria
    read_png_image(argv[1], &image, &width, &height, &image_size);

    // Processamento da imagem (alteracao do hue)

    // Versao sequencial:
    modify_hue_seq(image, width, height, hue_diff);

    // // Versao paralela
    // modify_hue(image, width, height, image_size, hue_diff);

    // Escrita da imagem para arquivo
    write_png_image(argv[2], image, width, height);

    // Liberacao de memoria
    free(image);
    return 0;
}
