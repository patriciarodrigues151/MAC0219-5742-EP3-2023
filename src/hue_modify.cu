// EP3 - MAC219 - Programação Concorrente e Paralela
// Integrantes:
// Patrícia Rodrigues, NUSP: 11315590
// Sabrina Araújo da Silva, NUSP: 12566182
// Samantha Miyahira, NUSP: 11797261

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <png.h>

// Funcao que aplica a matriz de transformacao A
// ao pixel px = (r, g, b)
// (new_r, new_g, new_b)' = A * (r, g, b)'
__host__ __device__ void modify_pixel(png_bytep px, double *A) {
    double r = px[0] / 255.0;
    double g = px[1] / 255.0;
    double b = px[2] / 255.0;

    double new_r = A[0] * r + A[1] * g + A[2] * b;
    double new_g = A[3] * r + A[4] * g + A[5] * b;
    double new_b = A[6] * r + A[7] * g + A[8] * b;

    new_r = fmin(fmax(new_r, 0.0), 1.0);
    new_g = fmin(fmax(new_g, 0.0), 1.0);
    new_b = fmin(fmax(new_b, 0.0), 1.0);

    px[0] = (png_byte) round(new_r * 255.0);
    px[1] = (png_byte) round(new_g * 255.0);
    px[2] = (png_byte) round(new_b * 255.0);
}

// Altera a matiz (hue) de uma imagem sequencialmente
void modify_hue_seq(png_bytep image, int width, int height, double hue_diff) {
    double c = cos(2 * M_PI * hue_diff);
    double s = sin(2 * M_PI * hue_diff);
    double one_third = 1.0 / 3.0;
    double sqrt_third = sqrt(one_third);

    // Matriz A compoe as operacoes de
    // conversao de RGB para HSV, mudanca de hue,
    // e conversao de HSV de volta para RGB
    // (new_r, new_g, new_b)' = A * (r, g, b)'
    // https://stackoverflow.com/questions/8507885/shift-hue-of-an-rgb-color

    double a11 = c + one_third * (1.0 - c);
    double a12 = one_third * (1.0 - c) - sqrt_third * s;
    double a13 = one_third * (1.0 - c) + sqrt_third * s;
    double a21 = a13; double a22 = a11; double a23 = a12;
    double a31 = a12; double a32 = a13; double a33 = a11;

    double A[9] = {a11, a12, a13, a21, a22, a23, a31, a32, a33};

    for (int i = 0; i < height; i++) {
        png_bytep row = &(image[i * width * 3]);
        for (int j = 0; j < width; j++) {
            png_bytep px = &(row[j * 3]);
            modify_pixel(px, A);
        }
    }
}

// Funcao auxiliar para identificar erros CUDA
void checkErrors(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "%s [Erro CUDA: %s]\n",
                msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// Kernel CUDA para alteracao do hue
// Voce deve modificar essa funcao no EP3
__global__ void modify_hue_kernel(png_bytep d_image, int width, int height, double *A) {
    // SEU CODIGO DO EP3 AQUI

    //calcula coordenadas do pixel para a thread CUDA
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    //caso a thread esteja nos limites válidos da imagem
    if(idx < width && idy < height){
        //indice do pixel no vetor 'd_image', 3 canais RBG
        int pixel_idx = (idy * width + idx) * 3;
        //valores são lidos e normalizados no  intervalo [0, 1], / 255
        double r = d_image[pixel_idx] / 255.0;
        double g = d_image[pixel_idx + 1] / 255.0;
        double b = d_image[pixel_idx + 2] / 255.0;

        //novos valores de RGB

        double novo_r = r * A[0] + g * A[1] + b * A[2];
        double novo_g = r * A[3] + g * A[4] + b * A[5];
        double novo_b = r * A[6] + g * A[7] + b * A[8];

        //min e max garantem que os novos valores estejam dentro do intervalo [0, 1]
        novo_r = fmin(fmax(novo_r, 0.0), 1.0);
        novo_g = fmin(fmax(novo_g, 0.0), 1.0);
        novo_b = fmin(fmax(novo_b, 0.0), 1.0);

        //após garantir que os valores estão no intervalo, os valores são normalizados de volta para o intervalo [0, 255] e arredondando para o valor inteiro mais proximo e armazenados na imagem de saída (d_image)
        d_image[pixel_idx] = (png_byte)round(novo_r * 255.0);
        d_image[pixel_idx + 1] = (png_byte)round(novo_g * 255.0);
        d_image[pixel_idx + 2] = (png_byte)round(novo_b * 255.0);
    }
}



// Altera a matiz (hue) de uma imagem em paralelo
// Voce deve modificar essa funcao no EP3
// Função para calcular a matriz A com base no desvio de matiz (hue_diff)

//SEU CODIGO AQUI
void calculate_A(double *A, double hue_diff) {
    double c = cos(2 * M_PI * hue_diff);
    double s = sin(2 * M_PI * hue_diff);
    double one_third = 1.0 / 3.0;
    double sqrt_third = sqrt(one_third);

    // Preenche a matriz A com os valores calculados a partir de hue_diff
    A[0] = c + one_third * (1.0 - c);
    A[1] = one_third * (1.0 - c) - sqrt_third * s;
    A[2] = one_third * (1.0 - c) + sqrt_third * s;
    A[3] = A[2];
    A[4] = A[0];
    A[5] = A[1];
    A[6] = A[1];
    A[7] = A[2];
    A[8] = A[0];
}

// Função para alocar memória no dispositivo (GPU) e copiar os dados
void allocate_and_copy(double *h_A, png_bytep h_image, int width, int height, size_t image_size, double **d_A, png_bytep *d_image) {
    // Aloca memória para a matriz A no dispositivo (GPU)
    hipMalloc((void **)d_A, sizeof(double) * 9);
    // Copia os dados da matriz A do host para o dispositivo
    hipMemcpy(*d_A, h_A, sizeof(double) * 9, hipMemcpyHostToDevice);

    // Aloca memória para a imagem de entrada no dispositivo (GPU)
    hipMalloc((void **)d_image, image_size);
    // Copia os dados da imagem do host para o dispositivo
    hipMemcpy(*d_image, h_image, image_size, hipMemcpyHostToDevice);
}

// Função para copiar a imagem modificada de volta para o host (CPU) e liberar memória no dispositivo (GPU)
void copy_back_and_free(png_bytep h_image, png_bytep d_image, size_t image_size) {
    // Copia a imagem modificada do dispositivo para o host
    hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
    // Libera a memória alocada no dispositivo
    hipFree(d_image);
}


// Função para modificar o matiz em paralelo usando CUDA
void modify_hue(png_bytep h_image, int width, int height, size_t image_size, double hue_diff) {
    double A[9];
    calculate_A(A, hue_diff);

    double *d_A;
    png_bytep d_image;
    allocate_and_copy(A, h_image, width, height, image_size, &d_A, &d_image);

    // Configura as dimensões do grid e blocos para a chamada do kernel
    dim3 dim_block(16, 16);
    dim3 dim_grid((width + dim_block.x - 1) / dim_block.x, (height + dim_block.y - 1) / dim_block.y);

    // Chama o kernel para modificar o matiz em paralelo
    modify_hue_kernel<<<dim_grid, dim_block>>>(d_image, width, height, d_A);
    hipDeviceSynchronize();

    // Copia a imagem modificada de volta para o host e libera a memória no dispositivo
    copy_back_and_free(h_image, d_image, image_size);
    hipFree(d_A);
}


// Le imagem png de um arquivo de entrada para a memoria
void read_png_image(const char *filename,
                    png_bytep *image,
                    int *width,
                    int *height,
                    size_t *image_size) {
    FILE *fp = fopen(filename, "rb");
    if (!fp) {
        fprintf(stderr, "Erro ao ler o arquivo de entrada %s\n", filename);
        exit(EXIT_FAILURE);
    }

    png_structp png = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png) {
        fprintf(stderr, "Erro ao criar PNG read struct \n");
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_infop info = png_create_info_struct(png);
    if (!info) {
        fprintf(stderr, "Erro ao criar PNG info struct \n");
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    // Em caso de erro nas funcoes da libpng,
    // programa "pula" para este ponto de execucao
    if (setjmp(png_jmpbuf(png))) {
        fprintf(stderr, "Erro ao ler imagem PNG \n");
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_init_io(png, fp);
    png_read_info(png, info);

    *width = png_get_image_width(png, info);
    *height = png_get_image_height(png, info);
    png_byte color_type = png_get_color_type(png, info);
    png_byte bit_depth = png_get_bit_depth(png, info);

    // Verifica se imagem png possui o formato apropriado
    if ((color_type != PNG_COLOR_TYPE_RGB && color_type != PNG_COLOR_TYPE_GRAY)
        || bit_depth != 8) {
        printf("Formato PNG nao suportado, deve ser 8-bit RGB ou grayscale\n");
        png_destroy_read_struct(&png, &info, NULL);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_read_update_info(png, info);

    // Alocacao de memoria para imagem e ponteiros para as linhas
    *image_size = png_get_rowbytes(png, info) * (*height);
    *image = (png_bytep) malloc(*image_size);

    png_bytep *row_pointers = (png_bytep *) malloc(sizeof(png_bytep) * (*height));
    for (int i = 0; i < *height; i++) {
        row_pointers[i] = *image + i * png_get_rowbytes(png, info);
    }

    // Leitura da imagem para a memoria
    png_read_image(png, row_pointers);

    // Finalizacao da leitura
    png_destroy_read_struct(&png, &info, NULL);
    fclose(fp);
    free(row_pointers);
}

// Escreve imagem png da memoria para um arquivo de saida
void write_png_image(const char *filename,
                     png_bytep image,
                     int width,
                     int height) {
    FILE *fp = fopen(filename, "wb");
    if (!fp) {
        fprintf(stderr, "Erro ao criar o arquivo de saida %s\n", filename);
        exit(EXIT_FAILURE);
    }

    png_structp png = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    if (!png) {
        fprintf(stderr, "Erro ao criar PNG write struct \n");
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    png_infop info = png_create_info_struct(png);
    if (!info) {
        fprintf(stderr, "Erro ao criar PNG info struct.\n");
        png_destroy_write_struct(&png, &info);
        fclose(fp);
        exit(EXIT_FAILURE);
    }

    // Em caso de erro nas funcoes da libpng,
    // programa "pula" para este ponto de execucao
    if (setjmp(png_jmpbuf(png))) {
        printf("Erro ao escrever imagem PNG \n");
        png_destroy_write_struct(&png, &info);
        fclose(fp);
        return;
    }

    png_init_io(png, fp);

    // Configura o formato da imagem a ser criada
    png_set_IHDR(
        png, info, width, height, 8, PNG_COLOR_TYPE_RGB, PNG_INTERLACE_NONE,
        PNG_COMPRESSION_TYPE_DEFAULT, PNG_FILTER_TYPE_DEFAULT
    );

    png_write_info(png, info);

    // Criacao de ponteiros para as linhas
    png_bytep row_pointers[height];
    for (int i = 0; i < height; i++) {
        row_pointers[i] = &(image[i * width * 3]);
    }

    // Escrita da imagem a partir da memoria
    png_write_image(png, row_pointers);
    png_write_end(png, NULL);

    // Finalizacao da escrita
    png_destroy_write_struct(&png, &info);
    fclose(fp);
}

int main(int argc, char *argv[]) {
    png_bytep image;
    int width, height;
    size_t image_size;

    // Leitura e validacao dos parametros de entrada
    if (argc != 4) {
        printf("Uso: ./hue_modify <input_file> <output_file> <hue_diff>\n");
        printf("0.0 <= hue_diff <= 1.0\n");
        exit(EXIT_FAILURE);
    }

    double hue_diff;
    int ret = sscanf(argv[3], "%lf", &hue_diff);
    if (ret == 0 || ret == EOF) {
        fprintf(stderr, "Erro ao ler hue_diff\n");
        exit(EXIT_FAILURE);
    }

    if (hue_diff < 0.0 || hue_diff > 1.0) {
        fprintf(stderr, "hue_diff deve ser entre 0.0 e 1.0\n");
        exit(EXIT_FAILURE);
    }

    // Leitura da imagem para memoria
    read_png_image(argv[1], &image, &width, &height, &image_size);

    // Processamento da imagem (alteracao do hue)

    // Versao sequencial:
    modify_hue_seq(image, width, height, hue_diff);

    // // Versao paralela
    // modify_hue(image, width, height, image_size, hue_diff);

    // Escrita da imagem para arquivo
    write_png_image(argv[2], image, width, height);

    // Liberacao de memoria
    free(image);
    return 0;
}
